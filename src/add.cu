
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// 1. kernel function
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20; // 1M elements
    
    float *x, *y;
    // 2. Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    
    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    // 2. Run kernel on 1M elements on the CPU
    add<<<1, 1>>>(N, x, y);

    // 2. Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;
    
    // 2. Free memory
    hipFree(x);
    hipFree(y);
    
    return 0;
}